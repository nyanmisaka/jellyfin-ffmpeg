
#include <hip/hip_runtime.h>
/*
 * Copyright (c) 2020 Yaroslav Pogrebnyak <yyyaroslav@gmail.com>
 *
 * This file is part of FFmpeg.
 *
 * FFmpeg is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation; either
 * version 2.1 of the License, or (at your option) any later version.
 *
 * FFmpeg is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with FFmpeg; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA
 */

template<typename T0, typename T1>
__inline__ __device__ void overlay_func(
    int x_position, int y_position,
    T0* main, int main_linesize,
    int main_adj_x, int main_offset,
    int main_depth, int main_shift,
    T1* overlay, int overlay_linesize,
    int overlay_w, int overlay_h,
    T1* overlay_alpha, int alpha_linesize,
    int alpha_adj_x, int alpha_adj_y)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= overlay_w + x_position ||
        y >= overlay_h + y_position ||
        x < x_position ||
        y < y_position ) {

        return;
    }

    int overlay_x = x - x_position;
    int overlay_y = y - y_position;

    float alpha = 1.0;
    if (alpha_linesize) {
        alpha = overlay_alpha[alpha_adj_x * overlay_x + alpha_adj_y * overlay_y * alpha_linesize] / 255.0f;
    }

    int main_pos = main_adj_x * x + y * (main_linesize / sizeof(*main)) + (main_adj_x > 1 ? main_offset : 0);
    if (main_depth > 8) {
        T0 overlay_res = (T0)(alpha * overlay[overlay_x + overlay_y * overlay_linesize]) << (main_depth - 8);
        T0 main_res = (T0)((1.0f - alpha) * (main[main_pos] >> main_shift));
        main[main_pos] = (T0)(overlay_res + main_res) << main_shift;
    } else {
        main[main_pos] = alpha * overlay[overlay_x + overlay_y * overlay_linesize] + (1.0f - alpha) * main[main_pos];
    }
}

extern "C" {

#define OVERLAY_VARIANT(NAME, TYPE0) \
__global__ void Overlay_Cuda_ ## NAME( \
    int x_position, int y_position, \
    TYPE0* main, int main_linesize, \
    int main_adj_x, int main_offset, \
    int main_depth, int main_shift, \
    unsigned char* overlay, int overlay_linesize, \
    int overlay_w, int overlay_h, \
    unsigned char* overlay_alpha, int alpha_linesize, \
    int alpha_adj_x, int alpha_adj_y) \
{ \
    overlay_func( \
        x_position, y_position, \
        main, main_linesize, \
        main_adj_x, main_offset, \
        main_depth, main_shift, \
        overlay, overlay_linesize, \
        overlay_w, overlay_h, \
        overlay_alpha, alpha_linesize, \
        alpha_adj_x, alpha_adj_y); \
}

OVERLAY_VARIANT(uchar, unsigned char)
OVERLAY_VARIANT(ushort, unsigned short)

}
