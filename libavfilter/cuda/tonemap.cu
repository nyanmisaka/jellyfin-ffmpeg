#include "hip/hip_runtime.h"
/*
 * This file is part of FFmpeg.
 *
 * FFmpeg is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation; either
 * version 2.1 of the License, or (at your option) any later version.
 *
 * FFmpeg is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with FFmpeg; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA
 */

#include "colorspace_common.h"
#include "pixfmt.h"
#include "tonemap.h"
#include "util.h"

extern __constant__ const enum TonemapAlgorithm tonemap_func;
extern __constant__ const float tone_param;
extern __constant__ const float desat_param;

#define mix(x, y, a) ((x) + ((y) - (x)) * (a))

static __inline__ __device__
float hable_f(float in) {
    float a = 0.15f, b = 0.50f, c = 0.10f, d = 0.20f, e = 0.02f, f = 0.30f;
    return (in * (in * a + b * c) + d * e) / (in * (in * a + b) + d * f) - e / f;
}

static __inline__ __device__
float direct(float s, float peak) {
    return s;
}

static __inline__ __device__
float linear(float s, float peak) {
    return s * tone_param / peak;
}

static __inline__ __device__
float gamma(float s, float peak) {
    float p = s > 0.05f ? s / peak : 0.05f / peak;
    float v = __powf(p, 1.0f / tone_param);
    return s > 0.05f ? v : (s * v / 0.05f);
}

static __inline__ __device__
float clip(float s, float peak) {
    return clamp(s * tone_param, 0.0f, 1.0f);
}

static __inline__ __device__
float reinhard(float s, float peak) {
    return s / (s + tone_param) * (peak + tone_param) / peak;
}

static __inline__ __device__
float hable(float s, float peak) {
    return hable_f(s) / hable_f(peak);
}

static __inline__ __device__
float mobius(float s, float peak) {
    float j = tone_param;
    float a, b;

    if (s <= j)
        return s;

    a = -j * j * (peak - 1.0f) / (j * j - 2.0f * j + peak);
    b = (j * j - 2.0f * j * peak + peak) / max(peak - 1.0f, FLOAT_EPS);

    return (b * b + 2.0f * b * j + j * j) / (b - a) * (s + a) / (s + b);
}

static __inline__ __device__
float bt2390(float s, float peak, float dst_peak) {
    float peak_pq = inverse_eotf_st2084(peak);
    float scale = peak_pq > 0.0f ? (1.0f / peak_pq) : 1.0f;

    float s_pq = inverse_eotf_st2084(s) * scale;
    float max_lum = inverse_eotf_st2084(dst_peak) * scale;

    float ks = 1.5f * max_lum - 0.5f;
    float tb = (s_pq - ks) / (1.0f - ks);
    float tb2 = tb * tb;
    float tb3 = tb2 * tb;
    float pb = (2.0f * tb3 - 3.0f * tb2 + 1.0f) * ks +
               (tb3 - 2.0f * tb2 + tb) * (1.0f - ks) +
               (-2.0f * tb3 + 3.0f * tb2) * max_lum;
    float sig = mix(pb, s_pq, s_pq < ks);

    return eotf_st2084(sig * peak_pq);
}

static __inline__ __device__
float map(float s, float peak, float dst_peak)
{
    switch (tonemap_func) {
    case TONEMAP_NONE:
    default:
        return direct(s, peak);
    case TONEMAP_LINEAR:
        return linear(s, peak);
    case TONEMAP_GAMMA:
        return gamma(s, peak);
    case TONEMAP_CLIP:
        return clip(s, peak);
    case TONEMAP_REINHARD:
        return reinhard(s, peak);
    case TONEMAP_HABLE:
        return hable(s, peak);
    case TONEMAP_MOBIUS:
        return mobius(s, peak);
    case TONEMAP_BT2390:
        return bt2390(s, peak, dst_peak);
    }
}

static __inline__ __device__
float3 map_one_pixel_rgb(float3 rgb, const FFCUDAFrame& src, const FFCUDAFrame& dst) {
    float sig = max(max(rgb.x, max(rgb.y, rgb.z)), FLOAT_EPS);
    float peak = src.peak;
    float dst_peak = dst.peak;

    // Rescale the variables in order to bring it into a representation where
    // 1.0 represents the dst_peak. This is because all of the tone mapping
    // algorithms are defined in such a way that they map to the range [0.0, 1.0].
    if (dst.peak > 1.0f) {
        sig *= 1.0f / dst.peak;
        peak *= 1.0f / dst.peak;
    }

    float sig_old = sig;

    // Desaturate the color using a coefficient dependent on the signal level
    if (desat_param > 0.0f) {
        float luma = get_luma_dst(rgb, luma_dst);
        float coeff = max(sig - 0.18f, FLOAT_EPS) / max(sig, FLOAT_EPS);
        coeff = __powf(coeff, 10.0f / desat_param);
        rgb = mix(rgb, make_float3(luma, luma, luma), make_float3(coeff, coeff, coeff));
    }

    sig = map(sig, peak, dst_peak);

    sig = min(sig, 1.0f);
    rgb = rgb * (sig / sig_old);
    return rgb;
}

// Map from source space YUV to destination space RGB
static __inline__ __device__
float3 map_to_dst_space_from_yuv(float3 yuv) {
    float3 c = yuv2lrgb(yuv);
    c = lrgb2lrgb(c);
    return c;
}

extern "C" {

__global__ void tonemap(FFCUDAFrame src, FFCUDAFrame dst)
{
    int xi = blockIdx.x * blockDim.x + threadIdx.x;
    int yi = blockIdx.y * blockDim.y + threadIdx.y;
    // each work item process four pixels
    int x = 2 * xi;
    int y = 2 * yi;

    if (y + 1 < src.height && x + 1 < src.width)
    {
        float3 yuv0 = read_px_flt(src, x,     y);
        float3 yuv1 = read_px_flt(src, x + 1, y);
        float3 yuv2 = read_px_flt(src, x,     y + 1);
        float3 yuv3 = read_px_flt(src, x + 1, y + 1);

        float3 c0 = map_to_dst_space_from_yuv(yuv0);
        float3 c1 = map_to_dst_space_from_yuv(yuv1);
        float3 c2 = map_to_dst_space_from_yuv(yuv2);
        float3 c3 = map_to_dst_space_from_yuv(yuv3);

        c0 = map_one_pixel_rgb(c0, src, dst);
        c1 = map_one_pixel_rgb(c1, src, dst);
        c2 = map_one_pixel_rgb(c2, src, dst);
        c3 = map_one_pixel_rgb(c3, src, dst);

        yuv0 = lrgb2yuv(c0);
        yuv1 = lrgb2yuv(c1);
        yuv2 = lrgb2yuv(c2);
        yuv3 = lrgb2yuv(c3);

        write_2x2_flt(dst, x, y, yuv0, yuv1, yuv2, yuv3);
    }
}

}
