#include "hip/hip_runtime.h"
/*
 * This file is part of FFmpeg.
 *
 * FFmpeg is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation; either
 * version 2.1 of the License, or (at your option) any later version.
 *
 * FFmpeg is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with FFmpeg; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA
 */

#include "colorspace_common.h"
#include "pixfmt.h"
#include "tonemap.h"
#include "util.h"

extern __constant__ const enum TonemapAlgorithm tonemap_func;
extern __constant__ const float tone_param;
extern __constant__ const float desat_param;
extern __constant__ const int enable_dither;
extern __constant__ const float dither_size;
extern __constant__ const float dither_quantization;

#define clamp(a, b, c) min(max((a), (b)), (c))
#define mix(x, y, a) ((x) + ((y) - (x)) * (a))
#define dot3(a, b) ((a).z * (b).z + ((a).y * (b).y + (a).x * (b).x))
#define dot4(a, b) ((a).w * (b).w + ((a).z * (b).z + ((a).y * (b).y + (a).x * (b).x)))

static __inline__ __device__
float get_dithered_y(float y, float d) {
    return floor(y * dither_quantization + d + 0.5f / (dither_size * dither_size)) * 1.0f / dither_quantization;
}

static __inline__ __device__
float hable_f(float in) {
    float a = 0.15f, b = 0.50f, c = 0.10f, d = 0.20f, e = 0.02f, f = 0.30f;
    return (in * (in * a + b * c) + d * e) / (in * (in * a + b) + d * f) - e / f;
}

static __inline__ __device__
float direct(float s, float peak) {
    return s;
}

static __inline__ __device__
float linear(float s, float peak) {
    return s * tone_param / peak;
}

static __inline__ __device__
float gamma(float s, float peak) {
    float p = s > 0.05f ? s / peak : 0.05f / peak;
    float v = __powf(p, 1.0f / tone_param);
    return s > 0.05f ? v : (s * v / 0.05f);
}

static __inline__ __device__
float clip(float s, float peak) {
    return clamp(s * tone_param, 0.0f, 1.0f);
}

static __inline__ __device__
float reinhard(float s, float peak) {
    return s / (s + tone_param) * (peak + tone_param) / peak;
}

static __inline__ __device__
float hable(float s, float peak) {
    return hable_f(s) / hable_f(peak);
}

static __inline__ __device__
float mobius(float s, float peak) {
    float j = tone_param;
    float a, b;

    if (s <= j)
        return s;

    a = -j * j * (peak - 1.0f) / (j * j - 2.0f * j + peak);
    b = (j * j - 2.0f * j * peak + peak) / max(peak - 1.0f, FLOAT_EPS);

    return (b * b + 2.0f * b * j + j * j) / (b - a) * (s + a) / (s + b);
}

static __inline__ __device__
float bt2390(float s, float peak, float dst_peak) {
    float peak_pq = inverse_eotf_st2084(peak);
    float scale = peak_pq > 0.0f ? (1.0f / peak_pq) : 1.0f;

    float s_pq = inverse_eotf_st2084(s) * scale;
    float max_lum = inverse_eotf_st2084(dst_peak) * scale;

    float ks = 1.5f * max_lum - 0.5f;
    float tb = (s_pq - ks) / (1.0f - ks);
    float tb2 = tb * tb;
    float tb3 = tb2 * tb;
    float pb = (2.0f * tb3 - 3.0f * tb2 + 1.0f) * ks +
               (tb3 - 2.0f * tb2 + tb) * (1.0f - ks) +
               (-2.0f * tb3 + 3.0f * tb2) * max_lum;
    float sig = mix(pb, s_pq, s_pq < ks);

    return eotf_st2084(sig * peak_pq);
}

static __inline__ __device__
float map(float s, float peak, float dst_peak)
{
    switch (tonemap_func) {
    case TONEMAP_NONE:
    default:
        return direct(s, peak);
    case TONEMAP_LINEAR:
        return linear(s, peak);
    case TONEMAP_GAMMA:
        return gamma(s, peak);
    case TONEMAP_CLIP:
        return clip(s, peak);
    case TONEMAP_REINHARD:
        return reinhard(s, peak);
    case TONEMAP_HABLE:
        return hable(s, peak);
    case TONEMAP_MOBIUS:
        return mobius(s, peak);
    case TONEMAP_BT2390:
        return bt2390(s, peak, dst_peak);
    }
}

static __inline__ __device__
float3 map_one_pixel_rgb_mode_max(float3 rgb, const FFCUDAFrame& src, const FFCUDAFrame& dst) {
    float sig = max(max(rgb.x, max(rgb.y, rgb.z)), FLOAT_EPS);
    float sig_old = sig;
    float peak = src.peak;
    float dst_peak = 1.0f;

    // Desaturate the color using a coefficient dependent on the signal level
    if (desat_param > 0.0f) {
        float luma = get_luma_dst(rgb, luma_dst);
        float coeff = max(sig - 0.18f, FLOAT_EPS) / max(sig, FLOAT_EPS);
        coeff = __powf(coeff, 10.0f / desat_param);
        rgb = mix(rgb, make_float3(luma, luma, luma), make_float3(coeff, coeff, coeff));
    }

    sig = map(sig, peak, dst_peak);
    sig = min(sig, 1.0f);
    rgb = rgb * (sig / sig_old);

    return rgb;
}

static __inline__ __device__
float3 map_one_pixel_rgb_mode_rgb(float3 rgb, const FFCUDAFrame& src, const FFCUDAFrame& dst) {
    float3 sig;
    sig.x = max(rgb.x, FLOAT_EPS);
    sig.y = max(rgb.y, FLOAT_EPS);
    sig.z = max(rgb.z, FLOAT_EPS);
    float3 sig_old = sig;
    float peak = src.peak;
    float dst_peak = 1.0f;

    // Desaturate the color using a coefficient dependent on the signal level
    if (desat_param > 0.0f) {
        float sig_max = max(max(rgb.x, max(rgb.y, rgb.z)), FLOAT_EPS);
        float luma = get_luma_dst(rgb, luma_dst);
        float coeff = max(sig_max - 0.18f, FLOAT_EPS) / max(sig_max, FLOAT_EPS);
        coeff = __powf(coeff, 10.0f / desat_param);
        rgb = mix(rgb, make_float3(luma, luma, luma), make_float3(coeff, coeff, coeff));
    }

    sig.x = map(sig.x, peak, dst_peak);
    sig.y = map(sig.y, peak, dst_peak);
    sig.z = map(sig.z, peak, dst_peak);
    sig.x = min(sig.x, 1.0f);
    sig.y = min(sig.y, 1.0f);
    sig.z = min(sig.z, 1.0f);
    rgb = rgb * (sig / sig_old);

    return rgb;
}

// Map from source space YUV to destination space RGB
static __inline__ __device__
float3 map_to_dst_space_from_yuv(float3 yuv) {
    float3 c = yuv2lrgb(yuv);
    return lrgb2lrgb(c);
}

static __inline__ __device__
float3 map_to_dst_space_from_yuv_dovi(float3 yuv) {
    float3 c = ycc2rgb(yuv.x, yuv.y, yuv.z);
    c = lms2rgb(c.x, c.y, c.z);
    return rgb2lrgb(c);
}

static __inline__ __device__
float3 map_to_dst_space_from_yuv_dovi_fast(float3 yuv) {
    float3 c = ycc2rgb(yuv.x, yuv.y, yuv.z);
    c = lms2rgb_fast(c.x, c.y, c.z);
    return rgb2lrgb(c);
}

static __inline__ __device__
float reshape_poly(float s, float4 coeffs) {
    return (coeffs.z * s + coeffs.y) * s + coeffs.x;
}

static __inline__ __device__
float reshape_mmr(float3 sig, float4 coeffs, float4 *dovi_mmr,
                  int dovi_mmr_single, int dovi_min_order, int dovi_max_order)
{
    int mmr_idx = dovi_mmr_single ? 0 : (int)coeffs.y;
    int order = (int)coeffs.w;
    float3 sigXxyz = make_float3(sig.x, sig.x, sig.y) * make_float3(sig.y, sig.z, sig.z);
    float4 sigX = make_float4(sigXxyz.x, sigXxyz.y, sigXxyz.z, sigXxyz.x * sig.z);
    float4 mmr;

    float s = coeffs.x;
    mmr = dovi_mmr[mmr_idx + 0];
    s += dot3(make_float3(mmr.x, mmr.y, mmr.z), sig);
    mmr = dovi_mmr[mmr_idx + 1];
    s += dot4(mmr, sigX);

    int t = dovi_max_order >= 2 && (dovi_min_order >= 2 || order >= 2);
    if (t) {
        float3 sig2 = sig * sig;
        float4 sigX2 = sigX * sigX;
        mmr = dovi_mmr[mmr_idx + 2];
        s += dot3(make_float3(mmr.x, mmr.y, mmr.z), sig2);
        mmr = dovi_mmr[mmr_idx + 3];
        s += dot4(mmr, sigX2);
        t = dovi_max_order == 3 && (dovi_min_order == 3 || order >= 3);
        if (t) {
            mmr = dovi_mmr[mmr_idx + 4];
            s += dot3(make_float3(mmr.x, mmr.y, mmr.z), sig2 * sig);
            mmr = dovi_mmr[mmr_idx + 5];
            s += dot4(mmr, sigX2 * sigX);
        }
    }

    return s;
}

static __inline__ __device__
float3 reshape_dovi_yuv(float3 yuv,
                        float *src_dovi_params, float *src_dovi_pivots,
                        float4 *src_dovi_coeffs, float4 *src_dovi_mmr)
{
    int i;
    float s;
    float3 sig = make_float3(clamp(yuv.x, 0.0f, 1.0f),
                             clamp(yuv.y, 0.0f, 1.0f),
                             clamp(yuv.z, 0.0f, 1.0f));
    float sig_arr[3] = {sig.x, sig.y, sig.z};
    float4 coeffs;
    int dovi_num_pivots, dovi_has_mmr, dovi_has_poly;
    int dovi_mmr_single, dovi_min_order, dovi_max_order;
    float dovi_lo, dovi_hi;
    float *dovi_params;
    float *dovi_pivots;
    float4 *dovi_coeffs, *dovi_mmr;

#pragma unroll
    for (i = 0; i < 3; i++) {
        dovi_params = src_dovi_params + i*8;
        dovi_pivots = src_dovi_pivots + i*8;
        dovi_coeffs = src_dovi_coeffs + i*8;
        dovi_mmr = src_dovi_mmr + i*48;
        dovi_num_pivots = dovi_params[0];
        dovi_has_mmr = dovi_params[1];
        dovi_has_poly = dovi_params[2];
        dovi_mmr_single = dovi_params[3];
        dovi_min_order = dovi_params[4];
        dovi_max_order = dovi_params[5];
        dovi_lo = dovi_params[6];
        dovi_hi = dovi_params[7];

        s = sig_arr[i];
        coeffs = dovi_coeffs[0];

        if (i == 0 && dovi_num_pivots > 2) {
            float t0 = s >= dovi_pivots[0], t1 = s >= dovi_pivots[1];
            float t2 = s >= dovi_pivots[2], t3 = s >= dovi_pivots[3];
            float t4 = s >= dovi_pivots[4], t5 = s >= dovi_pivots[5], t6 = s >= dovi_pivots[6];

            coeffs = mix(mix(mix(dovi_coeffs[0], dovi_coeffs[1], make_float4(t0, t0, t0, t0)),
                             mix(dovi_coeffs[2], dovi_coeffs[3], make_float4(t2, t2, t2, t2)),
                             make_float4(t1, t1, t1, t1)),
                         mix(mix(dovi_coeffs[4], dovi_coeffs[5], make_float4(t4, t4, t4, t4)),
                             mix(dovi_coeffs[6], dovi_coeffs[7], make_float4(t6, t6, t6, t6)),
                             make_float4(t5, t5, t5, t5)),
                         make_float4(t3, t3, t3, t3));
        }

        int has_mmr_poly = dovi_has_mmr && dovi_has_poly;

        if ((has_mmr_poly && coeffs.w == 0.0f) || (!has_mmr_poly && dovi_has_poly))
            s = reshape_poly(s, coeffs);
        else
            s = reshape_mmr(sig, coeffs, dovi_mmr,
                            dovi_mmr_single, dovi_min_order, dovi_max_order);

        sig_arr[i] = clamp(s, dovi_lo, dovi_hi);
    }

    return make_float3(sig_arr[0], sig_arr[1], sig_arr[2]);
}

extern "C" {

#define _READER \
    int xi = blockIdx.x * blockDim.x + threadIdx.x; \
    int yi = blockIdx.y * blockDim.y + threadIdx.y; \
    int x = 2 * xi; \
    int y = 2 * yi; \
    if (y + 1 >= src.height || x + 1 >= src.width) \
        return; \
    float3 yuv0 = read_tex_px_flt(src, x,     y); \
    float3 yuv1 = read_tex_px_flt(src, x + 1, y); \
    float3 yuv2 = read_tex_px_flt(src, x,     y + 1); \
    float3 yuv3 = read_tex_px_flt(src, x + 1, y + 1);

#define _RESHAPE \
    float *dovi_params = doviBuf; \
    float *dovi_pivots = doviBuf + 24; \
    float4 *dovi_coeffs = (float4 *)(doviBuf + 48); \
    float4 *dovi_mmr = (float4 *)(doviBuf + 144); \
    yuv0 = reshape_dovi_yuv(yuv0, dovi_params, dovi_pivots, dovi_coeffs, dovi_mmr); \
    yuv1 = reshape_dovi_yuv(yuv1, dovi_params, dovi_pivots, dovi_coeffs, dovi_mmr); \
    yuv2 = reshape_dovi_yuv(yuv2, dovi_params, dovi_pivots, dovi_coeffs, dovi_mmr); \
    yuv3 = reshape_dovi_yuv(yuv3, dovi_params, dovi_pivots, dovi_coeffs, dovi_mmr);

#define _YUV2RGB \
    float3 c0 = map_to_dst_space_from_yuv(yuv0); \
    float3 c1 = map_to_dst_space_from_yuv(yuv1); \
    float3 c2 = map_to_dst_space_from_yuv(yuv2); \
    float3 c3 = map_to_dst_space_from_yuv(yuv3);

#define _YCC2RGB \
    float3 c0 = map_to_dst_space_from_yuv_dovi(yuv0); \
    float3 c1 = map_to_dst_space_from_yuv_dovi(yuv1); \
    float3 c2 = map_to_dst_space_from_yuv_dovi(yuv2); \
    float3 c3 = map_to_dst_space_from_yuv_dovi(yuv3);

#define _YCC2RGB_F \
    float3 c0 = map_to_dst_space_from_yuv_dovi_fast(yuv0); \
    float3 c1 = map_to_dst_space_from_yuv_dovi_fast(yuv1); \
    float3 c2 = map_to_dst_space_from_yuv_dovi_fast(yuv2); \
    float3 c3 = map_to_dst_space_from_yuv_dovi_fast(yuv3);

#define _TONEMAP_MAX \
    c0 = map_one_pixel_rgb_mode_max(c0, src, dst); \
    c1 = map_one_pixel_rgb_mode_max(c1, src, dst); \
    c2 = map_one_pixel_rgb_mode_max(c2, src, dst); \
    c3 = map_one_pixel_rgb_mode_max(c3, src, dst);

#define _TONEMAP_RGB \
    c0 = map_one_pixel_rgb_mode_rgb(c0, src, dst); \
    c1 = map_one_pixel_rgb_mode_rgb(c1, src, dst); \
    c2 = map_one_pixel_rgb_mode_rgb(c2, src, dst); \
    c3 = map_one_pixel_rgb_mode_rgb(c3, src, dst);

#define _RGB2YUV \
    yuv0 = lrgb2yuv(c0); \
    yuv1 = lrgb2yuv(c1); \
    yuv2 = lrgb2yuv(c2); \
    yuv3 = lrgb2yuv(c3);

#define _DITHER \
    float d = read_dither(ditherTex, dither_size, xi, yi); \
    yuv0.x = get_dithered_y(yuv0.x, d); \
    yuv1.x = get_dithered_y(yuv1.x, d); \
    yuv2.x = get_dithered_y(yuv2.x, d); \
    yuv3.x = get_dithered_y(yuv3.x, d);

#define _WRITER \
    write_2x2_flt(dst, x, y, yuv0, yuv1, yuv2, yuv3);

#define TONEMAP_VARIANT(NAME, READER, RESHAPE, YUV2RGB, TONEMAP, RGB2YUV, DITHER, WRITER) \
__global__ void tonemap ## NAME( \
    FFCUDAFrame src, FFCUDAFrame dst, \
    hipTextureObject_t ditherTex, float *doviBuf) \
{ \
    READER \
    RESHAPE \
    YUV2RGB \
    TONEMAP \
    RGB2YUV \
    DITHER \
    WRITER \
}

TONEMAP_VARIANT(,              _READER,         , _YUV2RGB,   _TONEMAP_MAX, _RGB2YUV,        , _WRITER)
TONEMAP_VARIANT(_d,            _READER,         , _YUV2RGB,   _TONEMAP_MAX, _RGB2YUV, _DITHER, _WRITER)
TONEMAP_VARIANT(_rgb,          _READER,         , _YUV2RGB,   _TONEMAP_RGB, _RGB2YUV,        , _WRITER)
TONEMAP_VARIANT(_rgb_d,        _READER,         , _YUV2RGB,   _TONEMAP_RGB, _RGB2YUV, _DITHER, _WRITER)

TONEMAP_VARIANT(_dovi,         _READER, _RESHAPE, _YCC2RGB,   _TONEMAP_MAX, _RGB2YUV,        , _WRITER)
TONEMAP_VARIANT(_dovi_d,       _READER, _RESHAPE, _YCC2RGB,   _TONEMAP_MAX, _RGB2YUV, _DITHER, _WRITER)
TONEMAP_VARIANT(_dovi_rgb,     _READER, _RESHAPE, _YCC2RGB,   _TONEMAP_RGB, _RGB2YUV,        , _WRITER)
TONEMAP_VARIANT(_dovi_rgb_d,   _READER, _RESHAPE, _YCC2RGB,   _TONEMAP_RGB, _RGB2YUV, _DITHER, _WRITER)

TONEMAP_VARIANT(_dovi_f,       _READER, _RESHAPE, _YCC2RGB_F, _TONEMAP_MAX, _RGB2YUV,        , _WRITER)
TONEMAP_VARIANT(_dovi_d_f,     _READER, _RESHAPE, _YCC2RGB_F, _TONEMAP_MAX, _RGB2YUV, _DITHER, _WRITER)
TONEMAP_VARIANT(_dovi_rgb_f,   _READER, _RESHAPE, _YCC2RGB_F, _TONEMAP_RGB, _RGB2YUV,        , _WRITER)
TONEMAP_VARIANT(_dovi_rgb_d_f, _READER, _RESHAPE, _YCC2RGB_F, _TONEMAP_RGB, _RGB2YUV, _DITHER, _WRITER)

TONEMAP_VARIANT(_dovi_pq,      _READER, _RESHAPE, _YCC2RGB,               , _RGB2YUV,        , _WRITER)
TONEMAP_VARIANT(_dovi_pq_f,    _READER, _RESHAPE, _YCC2RGB_F,             , _RGB2YUV,        , _WRITER)

}
